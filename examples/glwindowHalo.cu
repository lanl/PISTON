#include "hip/hip_runtime.h"
/*
Copyright (c) 2011, Los Alamos National Security, LLC
All rights reserved.
Copyright 2011. Los Alamos National Security, LLC. This software was produced under U.S. Government contract DE-AC52-06NA25396 for Los Alamos National Laboratory (LANL),
which is operated by Los Alamos National Security, LLC for the U.S. Department of Energy. The U.S. Government has rights to use, reproduce, and distribute this software.

NEITHER THE GOVERNMENT NOR LOS ALAMOS NATIONAL SECURITY, LLC MAKES ANY WARRANTY, EXPRESS OR IMPLIED, OR ASSUMES ANY LIABILITY FOR THE USE OF THIS SOFTWARE.

If software is modified to produce derivative works, such modified software should be clearly marked, so as not to confuse it with the version available from LANL.

Additionally, redistribution and use in source and binary forms, with or without modification, are permitted provided that the following conditions are met:
·         Redistributions of source code must retain the above copyright notice, this list of conditions and the following disclaimer.
·         Redistributions in binary form must reproduce the above copyright notice, this list of conditions and the following disclaimer in the documentation and/or other
          materials provided with the distribution.
·         Neither the name of Los Alamos National Security, LLC, Los Alamos National Laboratory, LANL, the U.S. Government, nor the names of its contributors may be used
          to endorse or promote products derived from this software without specific prior written permission.

THIS SOFTWARE IS PROVIDED BY LOS ALAMOS NATIONAL SECURITY, LLC AND CONTRIBUTORS "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL LOS ALAMOS NATIONAL SECURITY, LLC OR CONTRIBUTORS BE LIABLE FOR ANY
DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA,
OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/

#ifdef __APPLE__
    #include <GL/glew.h>
    #include <OpenGL/OpenGL.h>
    #include <GLUT/glut.h>
#else
    #include <GL/glew.h>
    #include <GL/glut.h>
    #include <GL/gl.h>
#endif

#include <QtGui>
#include <QObject>


#ifdef USE_INTEROP
#include <cuda_gl_interop.h>
#endif

#include <piston/piston_math.h> 
#include <piston/choose_container.h>

#define SPACE thrust::detail::default_device_space_tag
using namespace piston;

#include <piston/halo_merge.h>   //wathsala

#include <sys/time.h>
#include <stdio.h>
#include <math.h>

#include "glwindowHalo.h"

#define STRINGIZE(x) #x
#define STRINGIZE_VALUE_OF(x) STRINGIZE(x)

struct timeval begin, end, diff;
int frame_count = 0;
int grid_size = 256;
float cameraFOV = 60.0;

halo *haloFinder;

// parameters needed for the halo_finder (look at halo_finder.h for definitions)
float linkLength, max_linkLength, min_linkLength;
int   particleSize, rL, np, n;

bool  haloFound, haloShow;
bool  particleSizeSelected, linkLengthSelected;
float step;

typedef thrust::tuple<float, float, float> Float3;
typedef thrust::device_vector<float>::iterator FloatIterator;
typedef thrust::tuple<FloatIterator, FloatIterator, FloatIterator> Float3IteratorTuple;
typedef thrust::zip_iterator<Float3IteratorTuple> Float3zipIterator;

typedef thrust::device_vector<int>::iterator   IntIterator;

thrust::host_vector<float3> vertices;
thrust::host_vector<float4> colors;

GLWindowHalo::GLWindowHalo(QWidget *parent) : QGLWidget(QGLFormat(QGL::SampleBuffers), parent)
{
  setFocusPolicy(Qt::StrongFocus);
  timer = new QTimer(this);
  connect(timer, SIGNAL(timeout()), this, SLOT(updateGL()));
  timer->start(1);
}

GLWindowHalo::~GLWindowHalo()
{

}

QSize GLWindowHalo::minimumSizeHint() const
{
  return QSize(100, 100);
}

QSize GLWindowHalo::sizeHint() const
{
  return QSize(1024, 1024);
}

bool GLWindowHalo::initialize(int argc, char *argv[])
{
  particleSizeSelected = false;
  linkLengthSelected   = true;
  haloFound = haloShow = false;
  step = 0.025;
  max_linkLength = 2;
  min_linkLength = 1;
  linkLength     = 1;
  particleSize   = 100;
  np = 256;
  rL = 64;
  n  = 1; //if you want a fraction of the file to load, use this.. 1/n

  char filename[1024];
  sprintf(filename, "%s/sub-24474", STRINGIZE_VALUE_OF(DATA_DIRECTORY));
  std::string format = "csv";
//    sprintf(filename, "%s/256", STRINGIZE_VALUE_OF(DATA_DIRECTORY));
//    std::string format = "cosmo";

  haloFinder = new halo_merge(min_linkLength, max_linkLength, true, filename, format, n, np, rL);

  return true;
}

void GLWindowHalo::initializeGL()
{
  qrot.set(0,0,0,1);

  // glClearColor(0.0f, 0.0f, 0.0f, 0.0f);
  glEnable(GL_DEPTH_TEST);
  glShadeModel(GL_SMOOTH);

  // good old-fashioned fixed function lighting
  float white[] = { 0.8, 0.8, 0.8, 1.0 };
  float black[] = { 0.0, 0.0, 0.0, 1.0 };
  float lightPos[] = { 0.0, 0.0, grid_size*1.5, 1.0 };

  glLightfv(GL_LIGHT0, GL_AMBIENT, white);
  glLightfv(GL_LIGHT0, GL_DIFFUSE, white);
  glLightfv(GL_LIGHT0, GL_SPECULAR, black);
  glLightfv(GL_LIGHT0, GL_POSITION, lightPos);

  glLightModeli(GL_LIGHT_MODEL_TWO_SIDE, 1);

  glEnable(GL_LIGHTING);
  glEnable(GL_LIGHT0);
  glEnable(GL_NORMALIZE);
  glColorMaterial(GL_FRONT_AND_BACK, GL_AMBIENT_AND_DIFFUSE);
  glEnable(GL_COLOR_MATERIAL);

  // Setup the view of the cube.
  glMatrixMode(GL_PROJECTION);
  gluPerspective( cameraFOV, 1.0, 1.0, grid_size*4.0);

  glMatrixMode(GL_MODELVIEW);
  glLoadIdentity();
  gluLookAt(0.0, 0.0, grid_size*1.5,
            0.0, 0.0, 0.0,
            0.0, 1.0, 0.0);

  glEnableClientState(GL_VERTEX_ARRAY);
  glEnableClientState(GL_COLOR_ARRAY);
}

struct tuple2float3 : public thrust::unary_function<Float3, float3>
{
  __host__ __device__
  float3 operator()(Float3 xyz)
  {
   return make_float3((float) thrust::get<0>((xyz)),
                      (float) thrust::get<1>((xyz)),
                      (float) thrust::get<2>((xyz)));
  }
};

struct setColor
{
  float4 *color;
  float *R, *G, *B;
	bool useF;

  __host__ __device__
  setColor(float4 *color, float *R, float *G, float *B, bool useF=false) :
      color(color), R(R), G(G), B(B), useF(useF) {}

  __host__ __device__
  void operator()(int i)
  {
    int haloIndU  = haloFinder->getHaloInd(i, useF);
    color[i] = make_float4(R[haloIndU],G[haloIndU],B[haloIndU],1);
  }
};

void GLWindowHalo::paintGL()
{
  timer->stop();

  if (frame_count == 0) gettimeofday(&begin, 0);

  glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);

  glMatrixMode(GL_PROJECTION);
  glLoadIdentity();
  gluPerspective( cameraFOV, 1.0, 1.0, grid_size*4.0);

  // set view matrix for 3D scene
  glMatrixMode(GL_MODELVIEW);
  glPushMatrix();

  qrot.getRotMat(rotationMatrix);
  glMultMatrixf(rotationMatrix);

  glTranslatef(-(grid_size-1)/2, -(grid_size-1)/2, -(grid_size-1)/2);

	if(haloFound && haloShow)
  {
		vertices.resize(haloFinder->numOfHaloParticles_f);
		colors.resize(haloFinder->numOfHaloParticles_f);

		thrust::copy(thrust::make_transform_iterator(haloFinder->vertices_begin_f(), tuple2float3()),
               thrust::make_transform_iterator(haloFinder->vertices_end_f(),   tuple2float3()),
               vertices.begin());

    thrust::for_each(CountingIterator(0), CountingIterator(0)+haloFinder->numOfHaloParticles_f,
        setColor(thrust::raw_pointer_cast(&*colors.begin()),
                 thrust::raw_pointer_cast(&*haloFinder->haloColorsR.begin()),
                 thrust::raw_pointer_cast(&*haloFinder->haloColorsG.begin()),
                 thrust::raw_pointer_cast(&*haloFinder->haloColorsB.begin()),
								 true));
  }
  else
  {
		vertices.resize(haloFinder->numOfParticles);
	  colors.resize(haloFinder->numOfParticles);

		thrust::copy(thrust::make_transform_iterator(haloFinder->vertices_begin(), tuple2float3()),
	               thrust::make_transform_iterator(haloFinder->vertices_end(),   tuple2float3()),
	               vertices.begin());

    thrust::fill(colors.begin(), colors.end(), make_float4(1,0,0,1));
  }

  glColorPointer(4, GL_FLOAT, 0, &colors[0]);
  glVertexPointer(3, GL_FLOAT, 0, &vertices[0]);
  glDrawArrays(GL_POINTS, 0, vertices.size());
  glPopMatrix();

  gettimeofday(&end, 0);
  timersub(&end, &begin, &diff);
  frame_count++;
  float seconds = diff.tv_sec + 1.0E-6*diff.tv_usec;
  if (seconds > 0.5f)
  {
    char title[256];
    sprintf(title, "Halo Finder, fps: %2.2f", float(frame_count)/seconds);
//      std::cout << title << std::endl;
    seconds = 0.0f;
    frame_count = 0;
  }

  timer->start(1);
}

void GLWindowHalo::resizeGL(int width, int height)
{
  glViewport(0, 0, width, height);
}


void GLWindowHalo::mousePressEvent(QMouseEvent *event)
{
  lastPos = event->pos();
}


void GLWindowHalo::mouseMoveEvent(QMouseEvent *event)
{
  int dx = event->x() - lastPos.x();
  int dy = event->y() - lastPos.y();

  if (event->buttons() & Qt::LeftButton)
  {
    Quaternion newRotX;
    newRotX.setEulerAngles(-0.2*dx*3.14159/180.0, 0.0, 0.0);
    qrot.mul(newRotX);

    Quaternion newRotY;
    newRotY.setEulerAngles(0.0, 0.0, -0.2*dy*3.14159/180.0);
    qrot.mul(newRotY);
  }
  else if (event->buttons() & Qt::RightButton)
  {
    cameraFOV += dy/20.0;
  }
  lastPos = event->pos();
}


void GLWindowHalo::keyPressEvent(QKeyEvent *event)
{
  //toggle between showing halos
  if ((event->key() == 'h') || (event->key() == 'H')) 
  {
    if (!haloShow && !haloFound)
    {
      (*haloFinder)(linkLength, particleSize);
      haloFound = true;
    }
    haloShow = !haloShow;
  }

  //toggle between changing linkLength & particleSize
  if ((event->key() == 't') || (event->key() == 'T')) 
  {
    particleSizeSelected = !particleSizeSelected;
    linkLengthSelected   = !linkLengthSelected;

    std::cout << (linkLengthSelected ? "linkLength Selected": "particleSize Selected") << std::endl;
  }

  if ((event->key() == '+') || (event->key() == '='))
  {
    if (linkLengthSelected) linkLength += step;
    else if (particleSizeSelected) particleSize += step;

    std::cout << "new input..." << std::endl;
    std::cout << "linkLength : " << linkLength << ", particleSize : " << particleSize << std::endl;

    (*haloFinder)(linkLength, particleSize);
  }
  else if ((event->key() == '-') || (event->key() == '_'))
  {
    if (linkLengthSelected) linkLength -= step;
    else if (particleSizeSelected) particleSize -= step;

    std::cout << "new input..." << std::endl;
    std::cout << "linkLength : " << linkLength << ", particleSize : " << particleSize << std::endl;

    (*haloFinder)(linkLength, particleSize);
  }
}

void GLWindowHalo::setLinkLength(int val)
{
	float ll = (max_ll-min_ll)*(val/100) + min_ll;

	linkLength = ll;
}

void GLWindowHalo::setParticleSize(int val)
{
	float pz = (200-0)*(val/100) + 0;

	particleSize = pz;
}


